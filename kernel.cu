﻿
#include "hip/hip_runtime.h"

#include <stdlib.h>

#include <stdio.h>
#include <math.h>
#include <time.h>


hipError_t addWithCuda(int *c,  int *a,  int *b, unsigned long long size);

__global__ 
void addKernel(int *c,  int *a,  int *b , unsigned long long size )
{

    int i = threadIdx.x+blockDim.x*blockIdx.x;
    if(i<size) c[i] = a[i] + b[i];
}

__host__
int main()
{
    clock_t start, end;
    double full_time; 
    
    long long  n = 200000000;
  
    int *a =(int*) malloc(sizeof(int) * n);
    int *b = (int*)malloc(sizeof(int) * n);
    int *c = (int*)malloc(sizeof(int) * n);

    if (c == NULL)
    {
        printf("Unable to allocate memory\n");
        exit(1);
    }

    for (int i = 0; i < n ; ++i)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
        c[i] = 0;
    }
    

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, n);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
  
    
    for (int i = 0; i < n; ++i)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
        c[i] = 0;
    }
    start = clock(); 

    for (int i = 0; i < n; ++i)
    {
        c[i] = b[i] + a[i];
    }
    end = clock(); 
    full_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("serial : %lf", full_time);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c,  int *a,  int *b, unsigned long long size)
{
    clock_t start, end;
    double full_time;
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;


    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! c");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        system("pause");
        fprintf(stderr, "hipMalloc failed! a");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! b");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! a");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! b ");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
    start = clock();
    addKernel<<<(long long) ceil((float)size/1024), 1024 >>>(dev_c, dev_a, dev_b, size);
    end = clock();
    full_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("parallel : %lf", full_time);
    


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
